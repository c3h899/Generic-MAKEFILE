#include "hip/hip_runtime.h"
#include<cstdio>
#include <omp.h> // OpenMP
#include <mpi.h> // MPI Interface

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
	// Stock CPP
	printf("Hello World from CPU (CPP)!\n");

	// MPI
	MPI_Init(NULL, NULL);
	int world_size; MPI_Comm_size(MPI_COMM_WORLD, &world_size);
	int world_rank; MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
	char processor_name[MPI_MAX_PROCESSOR_NAME]; int name_len;
	MPI_Get_processor_name(processor_name, &name_len);
	printf("Hello world from processor %s, rank %d out of %d processors\n",
		processor_name, world_rank, world_size);
	MPI_Finalize();

	// OpenMP*
#pragma omp parallel                   
{
    printf("Hello World... from thread = %d/%d\n", 
           omp_get_thread_num(), omp_get_max_threads() );
} 

	// CUDA
	cuda_hello<<<1,1>>>();
	hipDeviceSynchronize();

	return 0;
}
